/*
 * Copyright (C) Marlin.2024 Elias Frantar (elias.frantar@ist.ac.at)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#ifndef MARLIN_CUDA_KERNEL_CUH
#define MARLIN_CUDA_KERNEL_CUH

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <iostream>
#include <stdio.h>
#include<algorithm>

constexpr int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

// Instances of `Vec` are used to organize groups of >>registers<<, as needed for instance as inputs to tensor core
// operations. Consequently, all corresponding index accesses must be compile-time constants, which is why we
// extensively use `#pragma unroll` throughout the kernel code to guarantee this.
template <typename T, int n>
struct Vec {
  T elems[n];
  __device__ T& operator[](int i) {
    return elems[i];
  }
};

// Matrix fragments for tensor core instructions; their precise layout is documented here: 
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-floating-point-type
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>; // quantization scales
using FragZ = Vec<half2, 1>;
using I2 = Vec<int,2>;
using I2_2 = Vec<I2,2>;
// Predicated asynchronous global->shared copy; used for inputs A where we apply predication to handle batchsizes that
// are not multiples of 16.
__device__ inline void cp_async4_pred(void* smem_ptr, const void* glob_ptr, bool pred = true) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .pred p;\n"
    "   setp.ne.b32 p, %0, 0;\n"
    "   @p cp.async.cg.shared.global [%1], [%2], %3;\n"
    "}\n" :: "r"((int) pred), "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
}

// Asynchronous global->shared copy with a cache hint indicating that the values may be evicted immediately; used for
// quantized weights B, which are only accessed precisely once and should thus not pollute the L2 cache which we need
// for inputs A and outputs C. 
__device__ inline void cp_async4_stream(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n" 
    "   .reg .b64 p;\n"
    "   createpolicy.fractional.L2::evict_first.b64 p, 1.0;"
    "   cp.async.cg.shared.global.L2::cache_hint [%0], [%1], %2, p;\n"
    "}\n" :: "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
}

// Async copy fence.
__device__ inline void cp_async_fence() {
  asm volatile("cp.async.commit_group;\n" ::);
}

// Wait until at most `n` async copy stages are still pending.
template <int n>
__device__ inline void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n" :: "n"(n));
}

// m16n8k16 tensor core mma instruction with fp16 inputs and fp32 output/accumulation.
__device__ inline void mma(const FragA& a_frag, const FragB& frag_b, FragC& frag_c) {
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  float* c = reinterpret_cast<float*>(&frag_c);
  asm volatile(
    "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
    "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
    : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
    :  "r"(a[0]),  "r"(a[1]),  "r"(a[2]),  "r"(a[3]),  "r"(b[0]),  "r"(b[1]),
       "f"(c[0]),  "f"(c[1]),  "f"(c[2]),  "f"(c[3])
  );
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared memory, directly in tensor core layout.
__device__ inline void ldsm4(FragA& frag_a, const void* smem_ptr) {
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
    : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3]) : "r"(smem)
  );
}

// automatically recognize it in all cases. 
template <int lut>
__device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile(
    "lop3.b32 %0, %1, %2, %3, %4;\n"
    : "=r"(res) : "r"(a), "r"(b), "r"(c), "n"(lut)
  );
  return res;
}

__device__ inline FragB dequant_faster(int& q) {
  const int LO = 0x00070007;
  const int HI = 0x00380038;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  int hi = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  const int SUB = 0x64006400;
  const int MUL = 0x30003000;
  const int ADD = 0xd800d800;
  FragB frag_b;
  frag_b[0] = __hsub2(
    *reinterpret_cast<half2*>(&lo),
    *reinterpret_cast<const half2*>(&SUB)
  );
  frag_b[1] = __hfma2(
    *reinterpret_cast<half2*>(&hi),
    *reinterpret_cast<const half2*>(&MUL), *reinterpret_cast<const half2*>(&ADD)
  );
  return frag_b;
}

// Multiply dequantized values by the corresponding quantization scale; used only for grouped quantization.
__device__ inline void scale(FragB& frag_b, FragS& frag_s, int i, FragZ& frag_z) {
  half2 s = __half2half2(reinterpret_cast<__half*>(&frag_s)[i]);
  half2 z = __half2half2(reinterpret_cast<__half*>(&frag_z)[i]);
  frag_b[0] = __hfma2(frag_b[0], s, z);
  frag_b[1] = __hfma2(frag_b[1], s, z);
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int* lock, int count) {
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible globally.
      asm volatile ("ld.global.acquire.gpu.b32 %0, [%1];\n" : "=r"(state) : "l"(lock));
    while (state != count);
  }
  __syncthreads();
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int* lock, bool reset = false) {
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible globally, while releasing the barrier. 
    asm volatile ("fence.acq_rel.gpu;\n");
    asm volatile ("red.relaxed.gpu.global.add.s32 [%0], %1;\n" : : "l"(lock), "r"(val)); 
  }
}

template <
  const int threads, // number of threads in a threadblock
  const int thread_m_blocks, // number of 16x16 blocks in the m dimension (batchsize) of the threadblock 
  const int thread_n_blocks, // same for n dimension (output) 
  const int thread_k_blocks, // same for k dimension (reduction)
  const int stages, // number of stages for the async global->shared fetch pipeline
  const int group_blocks = 4 // number of consecutive 16x16 blocks with a separate quantization scale
>
__global__ void Marlin_3bit_with_zero(
  const int4* __restrict__ A, // fp16 input matrix of shape mxk 
  const int4* __restrict__ B1, // 3bit quantized weight matrix of shape kxn 
  const int4* __restrict__ B2,
        int4* __restrict__ C, // fp16 output buffer of shape mxn
  const int4* __restrict__ zero, // fp16 quantization zero points of shape (k/groupsize)xn 
  const int4* __restrict__ s, // fp16 quantization scales of shape (k/groupsize)xn 
  int  prob_m, // batch dimension m
  int  prob_n, // output dimension n
  int  prob_k, // reduction dimension k
  int* locks // extra global storage for barrier synchronization 
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the same size, which might involve multiple 
  // column "slices" (of width 16 * `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM example: 
  //   0 1 3 
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it ensures good utilization of all SMs
  // for many kinds of shape and GPU configurations, while requiring as few slow global cross-threadblock reductions as 
  // possible.
  
  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a better partitioning with less reductions
  //if( threadIdx.x == 0 & blockIdx.x == 0)
  //  printf("get s: %d, get s: %d, get s: %d, get s: %d", ((int*)s)[0], ((int*)s)[1], ((int*)s)[2], ((int*)s)[3]);

  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }
  //if (threadIdx.x == 0 && blockIdx.x == 0) printf("%d",zero==nullptr);

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);
  // Ensure that the number of tiles in each stripe is a multiple of the groupsize; this avoids an annoying special case
  // where a stripe starts in the middle of group.

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters; // number of threadblock tiles in the current slice
  int slice_count = 0; // total number of active threadblocks in the current slice
  int slice_idx; // index of threadblock in current slice; numbered bottom to top

  // We can easily implement parallel problem execution by just remapping indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for synchronization.
  auto init_slice = [&] () {
    slice_iters = iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row); //考虑到有的slice跨列
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles) 
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 8; // stride of the A matrix in global memory
  // We typically use `constexpr` to indicate that this value is a compile-time constant
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8; // stride of an A matrix tile in shared memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8; // delta between subsequent A tiles in global memory
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o); // between subsequent accesses within a tile
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o); // between shared memory writes
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4)); // between shared memory tile reads
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16; // within a shared memory tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks); // overall size of a tile
  constexpr int a_sh_wr_iters = ceildiv(a_sh_stage, a_sh_wr_delta); // number of shared write iterations for a tile

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta; //2

  int s_gl_stride = prob_n / 8; 
  int s_sh_stride = 16 * thread_n_blocks / 8;
  int s_sh_stage = s_sh_stride * ceildiv(thread_k_blocks,group_blocks);
  int s_gl_rd_delta = s_gl_stride * ceildiv(thread_k_blocks,group_blocks);
  int s_sh_rd_delta = 8 * (thread_n_blocks / 4) * (thread_k_blocks / b_sh_wr_iters / group_blocks);

  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);// Global A read index of current thread.
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);// Shared write index of current thread.
  int a_sh_rd = a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;// Shared read index.
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;

  int b_gl_rd_2 = b_gl_stride * ((threadIdx.x-32) / b_sh_stride) + ((threadIdx.x-32)% b_sh_stride);
  b_gl_rd_2 += b_sh_stride * slice_col;
  b_gl_rd_2 += b_gl_rd_delta_o * slice_row;
  
  int b_sh_wr = threadIdx.x; //threadid
  int b_sh_rd = threadIdx.x;//threadid
  bool B2_sh_wr_pred = b_sh_wr % 128 < 64 && b_sh_wr % 128 > 31;
  bool B1_sh_wr_pred = b_sh_wr % 64 < 32;
  int b1_sh_wr = (b_sh_wr / 64) * 32 + b_sh_wr % 64;
  int b2_sh_wr = (b_sh_wr / 128) * 32 + ((b_sh_wr - 32) % 128);
  bool B_sh_wr_pred = B1_sh_wr_pred || B2_sh_wr_pred;

  int s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) + s_sh_stage * slice_col + threadIdx.x;

  int s_sh_wr = threadIdx.x; //threadIdx.x
  int z_sh_wr = threadIdx.x - 32;
  int s_sh_rd;
  s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) / 4;

    // Precompute which thread should not read memory in which iterations; this is needed if there are more threads than
  // required for a certain tilesize or when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  //bool s_sh_wr_pred = threadIdx.x >= 96 && threadIdx.x <= 127;
  bool s_sh_wr_pred = threadIdx.x >=  96 && threadIdx.x < 128; //fetch to shared pred
  bool z_sh_wr_pred = threadIdx.x >= 224 && threadIdx.x < 256; //fetch to shared pred

  // To ensure that writing and reading A tiles to/from shared memory, the latter in fragment format, is fully bank
  // conflict free, we need to use a rather fancy XOR-based layout. The key here is that neither reads nor writes of 
  // the 16-byte `int4` blocks of 8 consecutive threads involve the same shared memory banks. Further, it seems (based
  // on NSight-Compute) that each warp must also write a consecutive memory segment?
  auto transform_a = [&] (int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };

  // Since the computation of this remapping is non-trivial and, due to our main loop unrolls, all shared memory 
  // accesses are static, we simply precompute both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
    #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] = transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at runtime; we break dependicies between
  // subsequent accesses with a tile by maintining multiple pointers (we have enough registers), a tiny optimization.
  const int4* B1_ptr[b_sh_wr_iters];
  const int4* B2_ptr[b_sh_wr_iters];

#pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
  {
    B1_ptr[i] = B1 + b_gl_rd_delta_i/2 * i+ (b_gl_rd / 64) * 32 + b_gl_rd % 64;;
    B2_ptr[i] = B2 + b_gl_rd_delta_i/4 * i+ (b_gl_rd_2 / 128) * 32 +  b_gl_rd_2 % 128; 
  }; 

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines. 
  int4* sh_a = sh;
  int4* sh_b1 = sh_a + stages * a_sh_stage;
  int4* sh_b2 = sh_b1 + stages * b_sh_stage/2;
  int4* sh_s = sh_b1 + stages * b_sh_stage;
  int4* sh_z = sh_s + stages * s_sh_stage;
  // Register storage for double buffer of shared memory reads. 
  FragA frag_a[2][thread_m_blocks]; //Vec<half2, 4>
  I2_2 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2]; //Vec<float,4> [4][2]
  FragS frag_s[2][4]; // Vec<half2, 1> [2][4]
  FragZ frag_z[2][4];
  // Zero accumulators.
  auto zero_accums = [&] () {
    #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next shared memory pipeline location.
  auto fetch_to_shared = [&] (int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
          &sh_a_stage[a_sh_wr_trans[i]],
          &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
          a_sh_wr_pred[i]
        );
      }   
      int4* sh_b1_stage = sh_b1 + (b_sh_stage/2) * pipe;
      int4* sh_b2_stage = sh_b2 + (b_sh_stage/4) * pipe; 
      int4* sh_s_stage = sh_s + s_sh_stage * pipe;
      int4* sh_z_stage = sh_z + s_sh_stage * pipe;   
      #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        int4* share_B = sh_b1_stage;
        const int4* B_ptr = B1_ptr[i];
        int4* sh_s_stage = sh_s + s_sh_stage * pipe;
        int4* sh_z_stage = sh_z + s_sh_stage * pipe;
        if (B1_sh_wr_pred){
          share_B = &sh_b1_stage[ (b_sh_wr_delta/2) * i + b1_sh_wr];
          B_ptr = B1_ptr[i];      
        }
        else if (B2_sh_wr_pred){
          share_B =&sh_b2_stage[ (b_sh_wr_delta/4) * i + b2_sh_wr];
          B_ptr = B2_ptr[i];
        }  
        cp_async4_pred(share_B, B_ptr, B_sh_wr_pred);
        B1_ptr[i] += b_gl_rd_delta_o/2;
        B2_ptr[i] += b_gl_rd_delta_o/4;
      }

      const int4* g = &s[s_gl_rd-96];
      int4* share = &sh_s_stage[s_sh_wr-96];
      if (z_sh_wr_pred){
        g = &zero[s_gl_rd-224];
        share = &sh_z_stage[s_sh_wr-224];
      }
      cp_async4_pred(share,g, s_sh_wr_pred|z_sh_wr_pred);
        //cp_async4_stream(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);*/

        //cp_async4_pred(&sh_z_stage[s_sh_wr-224], &zero[s_gl_rd-224], z_sh_wr_pred); 
      s_gl_rd += s_gl_rd_delta;
    }
    // Insert a fence even when we are winding down the pipeline to ensure that waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&] () {
    // We only have `stages - 2` active fetches since we are double buffering and can only issue the next fetch when
    // it is guaranteed that the previous shared memory load is fully complete (as it may otherwise be overwritten).; 
    cp_async_wait<stages - 2>();
    __syncthreads();
    
  };

  // Load the next sub-tile from the current location in the shared memory pipe into the current register buffer.
  auto fetch_to_registers = [&] (int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile; however, this does not seem to be a
    // significant bottleneck, while some theoretically better attempts have lead to bad instruction ordering by the
    // compiler and correspondingly a noticable drop in performance.
      //int4* sh_s_stage = sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) * (pipe / (group_blocks / thread_k_blocks)));
    int4* sh_s_stage = sh_s + s_sh_stage * pipe;
    reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd_delta * (k % b_sh_wr_iters) + s_sh_rd];
    int4* sh_z_stage = sh_z + s_sh_stage * pipe;
    reinterpret_cast<int4*>(&frag_z[k % 2])[0] = sh_z_stage[s_sh_rd_delta * (k % b_sh_wr_iters) + s_sh_rd];
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
    #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    I2* sh_b1_stage = reinterpret_cast<I2*>(sh_b1) + b_sh_stage * pipe;
    int* sh_b2_stage = reinterpret_cast<int*>(sh_b2) + b_sh_stage * pipe;
    frag_b_quant[k % 2][0] = sh_b1_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd];
    frag_b_quant[k % 2][1][0] = sh_b2_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd];
  };

  // Execute the actual tensor core matmul of a sub-tile. 
  auto matmul_faster = [&] (int k_mod_2) {
    int b_quant, b_quant_shift;
    int b_quant3 = 0;
    FragB frag_b0, frag_b1;
    #pragma unroll
    for (int j = 0; j < 3; j++) {
      b_quant = frag_b_quant[k_mod_2][j/2][j%2];
      b_quant_shift = b_quant >> 6;
      frag_b0 = dequant_faster(b_quant);
      // If there are no groups, we can just scale the final output once and can avoid doing so for each weight.
      scale(frag_b0, frag_s[k_mod_2][j], 0, frag_z[k_mod_2][j]);
      frag_b1 = dequant_faster(b_quant_shift);
      scale(frag_b1, frag_s[k_mod_2][j], 1, frag_z[k_mod_2][j]);
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k_mod_2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k_mod_2][i], frag_b1, frag_c[i][j][1]);
      }
      b_quant3 |= (b_quant& 0xf000f000) >> 4*(3-j);
    }   
    frag_b0 = dequant_faster(b_quant3);
    b_quant_shift = b_quant3 >> 6;
      // If there are no groups, we can just scale the final output once and can avoid doing so for each weight.
    scale(frag_b0, frag_s[k_mod_2][3], 0, frag_z[k_mod_2][3]);
    frag_b1 = dequant_faster(b_quant_shift);
    scale(frag_b1, frag_s[k_mod_2][3], 1, frag_z[k_mod_2][3]);
    //if(blockIdx.x == 0 && threadIdx.x == 0) printf("3,%x , %x, %x,%x, %x, %x \n:", b_quant, b_quant_shift,frag_b0[0],frag_b0[1],frag_b1[0],frag_b1[1]);    
    #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++) {
      mma(frag_a[k_mod_2][i], frag_b0, frag_c[i][3][0]);
      mma(frag_a[k_mod_2][i], frag_b1, frag_c[i][3][1]);
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the number of warps while keeping the n
  // dimension of a tile reasonable, we have multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&] () {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride; 
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
      // Parallel logarithmic shared memory reduction. We make sure to avoid any unnecessary read or write iterations,
      // e.g., for two warps we write only once by warp 1 and read only once by warp 0.       
      #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
        #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
            #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr = red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
                #pragma unroll
                for (int k = 0; k < 4; k++){
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] += c_rd[k] + c_wr[k];
                }
              }
              sh[red_sh_wr] = reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
          #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
            #pragma unroll
            for (int j = 0; j < 4; j++){
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] += c_rd[j];
            }
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we finally have to globally reduce over
  // the results. As the striped partioning minimizes the number of such reductions and our outputs are usually rather
  // small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&] (bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to maximize L2 cache utilization in this step. 
    // To do this, we write out results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) + 4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;
      int row = (threadIdx.x % 32) / 4;
      if (!first) {
        // Interestingly, doing direct global accesses here really seems to mess up the compiler and lead to slowdowns,
        // hence we also use async-copies even though these fetches are not actually asynchronous.
        #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
            &sh[c_sh_wr + c_sh_wr_delta * i],
            &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)],
            i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m
          );
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

      #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] += __half2float(
                reinterpret_cast<__half*>(&c_red)[j]
              );
            }
          }
          if (!last) {
            int4 c;
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<__half*>(&c)[j] = __float2half(
                reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]
              );
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] = c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually reshuffle matrix fragments in this step,
  // the reduction above is performed in fragment layout. 
  auto write_result = [&] () {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta = c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr = (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final global write patterns
    auto write = [&] (int idx, float c0, float c1, FragS& s) {
      half2 res = __halves2half2(__float2half(c0), __float2half(c1));
      ((half2*) sh)[idx] = res;
    };
    if (threadIdx.x / 32 < thread_n_blocks / 4) {
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0], frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2], frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0], frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2], frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

    
    #pragma unroll
    for (int i = 0; i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks)); i++) {      
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines. 
  auto start_pipes = [&] () {
    //printf("global to share \n");
    #pragma unroll
    for (int i = 0; i < stages - 1; i++)
      fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    //printf("share to register \n");
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();
  //int compute = 0, reduce = 0;
  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to ensure all shared memory accesses are
    // static. Note that both pipelines have even length meaning that the next iteration will always start at index 0.
   //clock_t start1 = clock();
    #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        //clock_t start1 = clock();
        fetch_to_registers(k + 1, pipe % stages);
        //clock_t end1 = clock();
        //register_time += end1 - start1;
        if (k == b_sh_wr_iters - 2) {
          //clock_t start2 = clock();
          fetch_to_shared((pipe + stages - 1) % stages, pipe, slice_iters >= stages);
          pipe++;
          wait_for_stage();
          //clock_t end2 = clock();
          //share_time += end2 - start2;
        }
        //clock_t start3 = clock();
        matmul_faster(k%2);
        //clock_t end3 = clock();
        //mma_time += end3 - start3;
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;
    
    // Process results and, if necessary, proceed to the next column slice. While this pattern may not be the most
    // readable, other ways of writing the loop seemed to noticeably worse performance after compliation.
   //clock_t end1 = clock();
    if (slice_iters == 0) {
      //clock_t end1 = clock();
      cp_async_wait<0>();
      //clock_t end2 = clock();
      //int cp_async = end2 - end1;
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before write-out
     //clock_t start1 = clock();
      thread_block_reduce();
      //clock_t end1 = clock();
      //int warp_reduce_time = end1-start1;
      //if (blockIdx.x == 0 && threadIdx.x == 0) printf("warp_reduce_time : %d \n", warp_reduce_time);
      
      //clock_t start1 = clock();
      if (slice_count > 1) { // only globally reduce if there is more than one block in a slice
        //printf("thread %d, block %d, use the global_reduce \n",threadIdx.x, blockIdx.x);
        //clock_t start1 = clock();
        barrier_acquire(&locks[slice_col], slice_idx);
        //clock_t start1 = clock();
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
        //clock_t end1 = clock();
        //int block_reduce_time = end1-start1;
        //if (blockIdx.x == 0 && threadIdx.x == 0) printf("block_reduce_time : %d \n", block_reduce_time);
      }
      //clock_t end1 = clock();
      //int block_reduce_time = end1-start1;
      //if (blockIdx.x == 0 && threadIdx.x == 0) printf("block_reduce_time : %d \n", block_reduce_time);


      if (last) // only the last block in a slice actually writes the result
      {
        //clock_t start1 = clock();

        //if(blockIdx.x == 0 && threadIdx.x == 0)printf("time statistics : %d, %d,  %d \n",register_time,share_time,mma_time);
        write_result();
        //clock_t end1 = clock();
        //int write_time = end1-start1;
        //if (blockIdx.x == 0 && threadIdx.x == 0) printf("write_time : %d \n", write_time);

      }
      //clock_t start1 = clock();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
        #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
        {
          B1_ptr[i] += (b_sh_stride - b_gl_rd_delta_o * k_tiles)/2;
          B2_ptr[i] += (b_sh_stride - b_gl_rd_delta_o * k_tiles)/4;
        }
          
        if (slice_col == 0) {
          #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++){
            B1_ptr[i] -= b_gl_stride/2;
            B2_ptr[i] -= b_gl_stride/4;
          }
        }
        s_gl_rd = s_sh_stage  * slice_col + threadIdx.x;
        start_pipes();
      }
      //clock_t end1 = clock();
      //int final = end1-start1;
      //if (blockIdx.x == 0 && threadIdx.x == 0) printf("final_time : %d \n", final);
    }
    //clock_t end2 = clock();
    //compute += end1-start1;
   //reduce += end2-end1;
    //if(blockIdx.x == 0 && threadIdx.x == 0) printf("compute time : , reduce_time: %d \n",reduce);
  //clock_t end1 = clock();
  //int outer_cycle = end1 - start1;
  //if(blockIdx.x == 0 && threadIdx.x == 0) printf("outer_cycle: %d \n",outer_cycle);
  }
  //if(blockIdx.x == 0 && threadIdx.x == 0) printf("compute time : , reduce_time: %d \n",reduce);
  
}


// 8 warps are a good choice since every SM has 4 schedulers and having more than 1 warp per schedule allows some more
// latency hiding. At the same time, we want relatively few warps to have many registers per warp and small tiles.
const int THREADS = 256;
const int STAGES = 4; // 4 pipeline stages fit into shared memory
const int SHARED_MEM = 96 * 1024; // max shared memory on compute capability 8.6 (< 8.0)

#define CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, GROUP_BLOCKS) \
  else if ( \
    thread_m_blocks == THREAD_M_BLOCKS && thread_n_blocks == THREAD_N_BLOCKS && thread_k_blocks == THREAD_K_BLOCKS && \
    group_blocks == GROUP_BLOCKS \
  ) { \
    hipFuncSetAttribute( \
      reinterpret_cast<const void*>(Marlin_3bit_with_zero<THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>), \
      hipFuncAttributeMaxDynamicSharedMemorySize, \
      SHARED_MEM \
    ); \
    Marlin_3bit_with_zero< \
      THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS \
    ><<<blocks, THREADS, SHARED_MEM, stream>>>( \
      A_ptr, B1_ptr, B2_ptr, C_ptr,  zero_ptr, s_ptr,\
      prob_m, prob_n, prob_k, \
      locks \
    ); \
  }

const int ERR_PROB_SHAPE = 1;
const int ERR_KERN_SHAPE = 2;

int marlin_cuda_3bit_with_zero(
  const void* A,
  const void* B1,
  const void* B2,
        void* C,
        void* s,
        void* zeros,
  int prob_m,
  int prob_n,
  int prob_k,
  void* workspace,
  int groupsize = -1,
  int dev = 0,
  hipStream_t stream = 0,
  int thread_k = -1,
  int thread_n = -1,
  int sms = -1,
  int max_par = 16
) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  if (thread_k == -1 || thread_n == -1) {
      thread_k = 64;
      thread_n = 256;
  }

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks =  groupsize / 16;
  //printf("here groupsize : %d", groupsize);
  int blocks = sms;
  if (group_blocks == 0)
    return ERR_PROB_SHAPE;
  if (prob_n % thread_n != 0 || prob_k % thread_k != 0 || (group_blocks != -1 && prob_k % group_blocks != 0))
    return ERR_PROB_SHAPE;
  if (prob_m == 0 || prob_n == 0 || prob_k == 0)
    return 0;

  const int4* A_ptr = (const int4*) A;
  const int4* B1_ptr = (const int4*) B1;
  const int4* B2_ptr = (const int4*) B2;

  int4* C_ptr = (int4*) C;
  const int4* s_ptr = (const int4*) s;
  const int4* zero_ptr = (const int4*) zeros;
  //std::cout << ((int*)zero_ptr)[2*256-1] << std::endl;
  int cols = prob_n / thread_n;
  int* locks = (int*) workspace;

  int ret = 0;
  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;

    if (thread_m_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any padding
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par)
        par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }
    //std::cout << ((int*)zero_ptr)[0] << std::endl;
    // For compilation speed, we only define the kernel configurations that have seemed useful (in terms of performance)
    // in our testing, however many more are, in principle, possible.
    if (false) {}
    CALL_IF(1, 16, 4, 4)
    CALL_IF(2, 16, 4, 4)
    CALL_IF(3, 16, 4, 4)
    CALL_IF(4, 16, 4, 4)
    else
      ret = ERR_KERN_SHAPE;

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }

  return ret;
}

#endif
